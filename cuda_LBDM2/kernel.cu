#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <>

#include <hipblas.h>
#include <hipsparse.h>

#include <helper_string.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

using namespace std;

#define len1  1
#define len3  3
#define len33  9
#define len4  4
#define len9  9
#define len93  27
#define len94  36
#define len99  81

#define lambda 0.000001
#define TINY 1.0e-40

#define a(i,j) a[(i)*len9+(j)]

#define channel  3

//=========================================================================================================================================================//

//simple matrix operation
__device__ void rowFirst_printMat(double *m, int rows, int cols)
{
	for (int j = 0; j<rows; ++j){
		for (int i = 0; i<cols; ++i){
			//printf("%.6f\t", m[i + j*cols]);
		}
		//printf("\n");
	}
}

__device__ void rowFirst_mul(double *A, double *B, int m, int n, int k, double *C)
{
	double sum;
	int ia, ib;
	for (int i = 0; i<m; ++i){
		for (int j = 0; j<n; ++j){
			sum = 0;
			for (int t = 0; t<k; ++t){
				ia = t + j*m;
				ib = i + t*k;
				sum += A[ia] * B[ib];
				//sum += A[j + t*m] * B[t + i*n];
			}
			C[i + j*m] = sum;
		}
	}
}

__device__ void rowFirst_mul_opt_94(double *Xi, double *A, double *B)
{
	double sum;
	int ia, ib;
	for (int j = 0; j<len9; ++j)
	{
		for (int i = 0; i<len9; ++i)
		{
			sum = 0;
			sum =
				Xi[j * len4 + 0] * Xi[i * len4 + 0] +
				Xi[j * len4 + 1] * Xi[i * len4 + 1] +
				Xi[j * len4 + 2] * Xi[i * len4 + 2] +
				Xi[j * len4 + 3] * Xi[i * len4 + 3];

			B[i + j*len9] = sum;
			A[i + j*len9] = sum;
		}
		A[j + j*len9] += lambda;
	}
	A[len99 - 1] -= lambda;
}

__device__ void rowFirst_lapcoeff(double *I_F, double *lapcoeff)
{
	double sum;
	int ia, ib;
	for (int j = 0; j<len9; ++j){
		for (int i = 0; i<len9; ++i){
			sum = 0;
			for (int t = 0; t<len9; ++t){
				ia = j + t*len9;
				ib = i + t*len9;
				sum += I_F[ia] * I_F[ib];
			}
			lapcoeff[i + j*len9] = sum;
		}
	}
}

__device__ void rowFirst_cal_I_F(double *F)
{
	for (int i = 0; i<len99; ++i)
		F[i] = -F[i];
	for (int i = 0; i<len9; ++i)
		F[i + i * len9] += 1;
}

//============================================================================//

//lu based solver
__device__ void Doolittle(int d, double*S, double*D){
	for (int k = 0; k<d; ++k){
		for (int j = k; j<d; ++j){
			double sum = 0.;
			for (int p = 0; p<k; ++p)sum += D[k*d + p] * D[p*d + j];
			D[k*d + j] = (S[k*d + j] - sum); // not dividing by diagonals
		}
		for (int i = k + 1; i<d; ++i){
			double sum = 0.;
			for (int p = 0; p<k; ++p)sum += D[i*d + p] * D[p*d + k];
			D[i*d + k] = (S[i*d + k] - sum) / D[k*d + k];
		}
	}
}
__device__ void solveDoolittle(int d, double*LU, double*b, double*x){
	double y[len9];
	for (int i = 0; i<d; ++i){
		double sum = 0.;
		for (int k = 0; k<i; ++k)sum += LU[i*d + k] * y[k];
		y[i] = (b[i] - sum); // not dividing by diagonals
	}
	for (int i = d - 1; i >= 0; --i){
		double sum = 0.;
		for (int k = i + 1; k<d; ++k)sum += LU[i*d + k] * x[k];
		x[i] = (y[i] - sum) / LU[i*d + i];
	}
}

__device__ void coutMatrix(int d, double*m){
	//printf("\n");
	for (int i = 0; i<d; ++i){
		for (int j = 0; j<d; ++j)
			//printf("%.4f", m[i*d + j]);
		//printf("\n");
	}
}
__device__ void coutVector(int d, double*v){
	//printf("\n");
	for (int j = 0; j<d; ++j)
		//printf("%.4f", v[j]);
	//printf("\n");
}


//=========================================================================================================================================================//

//helper function
__device__ void d_printD(double *data, int rows, int cols){
	//
	//printf("\n");

	for (int j = 0; j < rows; ++j)
	{
		for (int i = 0; i < cols; ++i)
		{
			//printf("%.3f\t", data[j + i*rows]);
		}
		//printf("\n");
	}
}
__device__ void d_printI(int *data, int rows, int cols){
	//
	//printf("\n");

	for (int j = 0; j < rows; ++j)
	{
		for (int i = 0; i < cols; ++i)
		{
			//printf("%d\t", data[j + i*rows]);
		}
		//printf("\n");
	}
}


__device__ void rowFirst_assignXiMatrix(double *imdata, double *Xi, int tid, int cols, int x, int y)
{
	Xi[0] = imdata[(tid - cols - 1) * 3 + 0];		Xi[1] = imdata[(tid - cols - 1) * 3 + 1];	Xi[2] = imdata[(tid - cols - 1) * 3 + 2];	Xi[3] = 1;
	Xi[4] = imdata[(tid - 1) * 3 + 0];				Xi[5] = imdata[(tid - 1) * 3 + 1];			Xi[6] = imdata[(tid - 1) * 3 + 2];			Xi[7] = 1;
	Xi[8] = imdata[(tid + cols - 1) * 3 + 0];		Xi[9] = imdata[(tid + cols - 1) * 3 + 1];	Xi[10] = imdata[(tid + cols - 1) * 3 + 2];	Xi[11] = 1;



	Xi[12] = imdata[(tid - cols + 0) * 3 + 0];		Xi[13] = imdata[(tid - cols + 0) * 3 + 1];	Xi[14] = imdata[(tid - cols + 0) * 3 + 2];	Xi[15] = 1;
	Xi[16] = imdata[(tid + 0) * 3 + 0];				Xi[17] = imdata[(tid + 0) * 3 + 1];			Xi[18] = imdata[(tid + 0) * 3 + 2];			Xi[19] = 1;
	Xi[20] = imdata[(tid + cols + 0) * 3 + 0];		Xi[21] = imdata[(tid + cols + 0) * 3 + 1];	Xi[22] = imdata[(tid + cols + 0) * 3 + 2];	Xi[23] = 1;



	Xi[24] = imdata[(tid - cols + 1) * 3 + 0];		Xi[25] = imdata[(tid - cols + 1) * 3 + 1];	Xi[26] = imdata[(tid - cols + 1) * 3 + 2];	Xi[27] = 1;
	Xi[28] = imdata[(tid + 1) * 3 + 0];				Xi[29] = imdata[(tid + 1) * 3 + 1];     	Xi[30] = imdata[(tid + 1) * 3 + 2];	        Xi[31] = 1;
	Xi[32] = imdata[(tid + cols + 1) * 3 + 0];		Xi[33] = imdata[(tid + cols + 1) * 3 + 1];	Xi[34] = imdata[(tid + cols + 1) * 3 + 2];	Xi[35] = 1;

}

__device__ void mapDeviceMem(
	double *d_Xi, double *d_A, double *d_B,
	int *d_trimap, int *d_row_inds, int *d_col_inds, double *d_vals,
 	//
	double *&Xi, double *&A, double *&B,
	int *&row_inds, int *&col_inds, double *&vals,
	int tid)
{
	Xi = &d_Xi[tid*len94];

	A = &d_A[tid*len99];
	B = &d_B[tid*len99];

	vals	 = &d_vals[tid*len99];
	row_inds = &d_row_inds[tid*len99];
	col_inds = &d_col_inds[tid*len99];
}


__device__ void compLapcoeff_new(double *Xi, double *A, double *B)
{

	rowFirst_mul_opt_94(Xi, A, B);
	Doolittle(len9, A, A);

	for (int i = 0; i < len9; ++i)
	{
		solveDoolittle(len9, A, &B[len9*i], &B[len9*i]);
	}

	rowFirst_cal_I_F(B);

	rowFirst_lapcoeff(B, A);

	for (int i = 0; i < len99; ++i)
		B[i] = A[i];

}

__global__ void kernel_new(double *d_Xi, double *d_A, double *d_B,
	double *d_imdata, int *d_trimap, int *d_row_inds, int *d_col_inds, double *d_vals,
	int rows, int cols)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int tid = x + y * cols;

	////printf("in the kernel\n");

	if ((x <= 1) || (y <= 1) || (x >= (cols - 1) ) || (y >= (rows - 1) ) || d_trimap[tid])	return;	


	double *Xi, *A, *B;

	int *row_inds, *col_inds;
	double *vals;


	mapDeviceMem(
		d_Xi, d_A, d_B, 
		d_trimap, d_row_inds, d_col_inds, d_vals,
		Xi, A, B,
		row_inds, col_inds, vals,
		tid);

	rowFirst_assignXiMatrix(d_imdata, Xi, tid, cols, x, y);

	compLapcoeff_new(Xi, A, B);

	//
	int winInds[9];
	winInds[0] = (y - 1) +	(x - 1)*rows;	winInds[3] = (y - 1) + (x    )*rows;	winInds[6] = (y - 1) + (x + 1)*rows;
	winInds[1] = (y    ) +	(x - 1)*rows;	winInds[4] = (y    ) + (x    )*rows;	winInds[7] = (y    ) + (x + 1)*rows;
	winInds[2] = (y + 1) +	(x - 1)*rows;	winInds[5] = (y + 1) + (x    )*rows;	winInds[8] = (y + 1) + (x + 1)*rows;

	//row
	for (int j = 0; j < len9; ++j)
		for (int i = 0; i < len9; ++i)
			row_inds[j*len9 + i] = winInds[i];

	//col
	for (int j = 0; j < len9; ++j)
		for (int i = 0; i < len9; ++i)
			col_inds[j*len9 + i] = winInds[j];

	//vals
	for (int i = 0; i < len99; ++i)
		vals[i] = B[i];

}

//=========================================================================================================================================================//

//memory allocation and free
void allocMem(
	int pixNum,
	double *&d_Xi, double *&d_A, double *&d_B,
	double *&d_imdata, double *&d_vals, int *&d_trimap, int *&d_row_inds, int *&d_col_inds)
{
	//d_Xi
	checkCudaErrors(hipMalloc((void**)&d_Xi, pixNum * len94 * sizeof(double)));

	//d_A
	checkCudaErrors(hipMalloc((void**)&d_A, pixNum * len99 * sizeof(double)));

	//d_B
	checkCudaErrors(hipMalloc((void**)&d_B, pixNum * len99 * sizeof(double)));

	//d_imdata
	checkCudaErrors(hipMalloc((void**)&d_imdata, pixNum * channel * sizeof(double)));

	//d_trimap
	checkCudaErrors(hipMalloc((void**)&d_trimap, pixNum * sizeof(int)));

	//d_vals
	checkCudaErrors(hipMalloc((void**)&d_vals, pixNum * len99 * sizeof(double)));
	
	//d_row_inds, d_col_inds
	checkCudaErrors(hipMalloc((void**)&d_row_inds, pixNum * len99 * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&d_col_inds, pixNum * len99 * sizeof(int)));
}

void freeMem(double *&d_Xi, double *&d_A, double *&d_B,
	double *&d_imdata, double *&d_vals, int *&d_trimap, int *&d_row_inds, int *&d_col_inds)
{
	hipFree(d_Xi);
	hipFree(d_A);
	hipFree(d_B);
	
	//
	hipFree(d_imdata);
	hipFree(d_vals);
	hipFree(d_trimap);
	hipFree(d_row_inds);
	hipFree(d_col_inds);

}

void printMatrix(double* m, int rows, int cols){
	//
	cout.precision(3);
	cout << endl;
	for (int j = 0; j < rows; ++j){
		for (int i = 0; i < cols; ++i)
		{
			cout << m[j + i*rows] << "\t";
		}
		cout << endl;
	}
}

//=========================================================================================================================================================//

//interface
extern "C" void callKernel_step1(int rows, int cols, double *imdata, int *trimap, int *row_inds, int *col_inds, double *vals)
{
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	
	int pixNum = rows * cols;

	double *d_Xi, *d_A, *d_B;

	double *d_imdata, *d_vals;
	int *d_trimap, *d_row_inds, *d_col_inds;

	//
	allocMem(pixNum, 
		d_Xi, d_A, d_B,
		d_imdata, d_vals, d_trimap, d_row_inds, d_col_inds);

	checkCudaErrors(hipMemcpy(d_imdata, imdata, pixNum * channel *sizeof(double), hipMemcpyHostToDevice ));
	checkCudaErrors(hipMemcpy(d_trimap, trimap, pixNum * sizeof(int), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemset(d_row_inds, 0, pixNum * len99 * sizeof(int)));
	checkCudaErrors(hipMemset(d_col_inds, 0, pixNum * len99 * sizeof(int)));
	checkCudaErrors(hipMemset(d_vals,	   0, pixNum * len99 * sizeof(double)));
	

	//dim3 block(16, 16);
	//dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);


	checkCudaErrors(hipEventRecord(start, 0));
	//kernel_new<<<grid, block>>>(d_Xi, d_A, d_B,
	//	d_imdata, d_trimap, d_row_inds, d_col_inds, d_vals,
	//	rows, cols);


	dim3 grid(1024, 1024);
	kernel_new<<<grid, 1>>>(d_Xi, d_A, d_B,
		d_imdata, d_trimap, d_row_inds, d_col_inds, d_vals,
		rows, cols);

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	float timeCost = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&timeCost, start, stop));
	//printf("time consumption on GPU: %f\n", timeCost);
	
	checkCudaErrors(hipMemcpy(row_inds, d_row_inds, pixNum * len99 * sizeof(int),		hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(col_inds, d_col_inds, pixNum * len99 * sizeof(int),		hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(vals,     d_vals,     pixNum * len99 * sizeof(double),	hipMemcpyDeviceToHost));


	freeMem(d_Xi, d_A, d_B,
		d_imdata, d_vals, d_trimap, d_row_inds, d_col_inds);
		
	
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	return ;

}

extern "C" void callKernel_step2(int *L_rows, int *L_cols, double *L_vals, double *alpha_star, double *res, int m, int nnz)
{
	hipsparseHandle_t handle;
	hipsparseCreate(&handle);
	hipsparseStatus_t status;

	double *d_csrVal;
	int *d_csrRowPtr, *d_csrColInd;

	double *h_x, *h_y, *h_z,
		   *d_x, *d_y, *d_z;

	hipsparseMatDescr_t descr_M = 0;
	hipsparseMatDescr_t descr_L = 0;
	hipsparseMatDescr_t descr_U = 0;
	csrilu02Info_t	info_M = 0;
	csrsv2Info_t	info_L = 0;
	csrsv2Info_t	info_U = 0;
	int pBufferSize_M;
	int pBufferSize_L;
	int pBufferSize_U;
	int pBufferSize;
	char *pBuffer = 0;
	int structural_zero;
	int numerical_zero;
	const double alpha = 1.;
	const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
	const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
	const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
	const hipsparseOperation_t trans_L = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	const hipsparseOperation_t trans_U = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	
	//=====================================================================================================//

	//matrix device
	int * csrRowPtr = 0;
	int * cooRowIndex = 0;
	int * cooColIndex = 0;
	double * cooVal = 0;

	/* allocate GPU memory and copy the matrix and vectors into it */
	checkCudaErrors(hipMalloc((void**)&cooRowIndex, nnz*sizeof(cooRowIndex[0])));
	checkCudaErrors(hipMalloc((void**)&cooColIndex, nnz*sizeof(cooColIndex[0])));
	checkCudaErrors(hipMalloc((void**)&cooVal, nnz*sizeof(cooVal[0])));

	checkCudaErrors(hipMalloc((void**)&d_x, m * sizeof(d_x[0])));
	checkCudaErrors(hipMalloc((void**)&d_y, m * sizeof(d_y[0])));
	checkCudaErrors(hipMalloc((void**)&d_z, m * sizeof(d_z[0])));
	//printf("Device malloc succeeded\n");

	//copy stage
	checkCudaErrors(hipMemcpy(cooRowIndex, L_rows, nnz*sizeof(L_rows[0]), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(cooColIndex, L_cols, nnz*sizeof(L_cols[0]), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(cooVal, L_vals, nnz*sizeof(L_vals[0]), hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMemcpy(d_x, alpha_star, m * sizeof(d_x[0]), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpy(d_y, h_y, m * sizeof(d_y[0]), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpy(d_z, h_z, m * sizeof(d_z[0]), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemset(d_y, 0, m * sizeof(d_y[0])));
	checkCudaErrors(hipMemset(d_z, 0, m * sizeof(d_z[0])));
	//printf("Memcpy from Host to Device succeeded\n");

	//change matrix format
	checkCudaErrors(hipMalloc((void**)&csrRowPtr, (m+1) * sizeof(csrRowPtr[0])));
	checkCudaErrors(hipsparseXcoo2csr(handle, cooRowIndex, nnz, m+1, csrRowPtr, HIPSPARSE_INDEX_BASE_ZERO));
	//printf("Conversion from COO to CSR format succeeded\n");

	hipDeviceSynchronize();


	////printf("the value is~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~:\n");
	//for (int i = 0; i < nnnz; ++i)
	//{
	//	////printf("i: %d row: %d\tcolumn: %d\tvalue:%f\n", i, L_rows[i], L_cols[i], L_vals[i]);
	//	//printf("%d\t%d\t%f\n", L_rows[i], L_cols[i], L_vals[i]);
	//}

	////printf("matrix size: %d\n", m);
	//for (int i = 0; i < m; ++i)
	//{
	//	////printf("i: %d as value:%f\n", i, alpha_star[i]);
	//	//printf("%f\n", alpha_star[i]);
	//}
	
	d_csrVal = cooVal;
	d_csrRowPtr = csrRowPtr;
	d_csrColInd = cooColIndex;

	//=====================================================================================================//

	hipsparseCreateMatDescr(&descr_M);
	hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);


	hipsparseCreateMatDescr(&descr_L);
	hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

	hipsparseCreateMatDescr(&descr_U);
	hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

	// step 2: create a empty info structure 
	// we need one info for csrilu02 and two info's for csrsv2
	hipsparseCreateCsrilu02Info(&info_M);
	hipsparseCreateCsrsv2Info(&info_L);
	hipsparseCreateCsrsv2Info(&info_U);

	// step 3: query how much memory used in csrilu02 and csrsv2, and allocate the buffer 
	hipsparseDcsrilu02_bufferSize(handle, m, nnz, descr_M, d_csrVal, d_csrRowPtr, d_csrColInd, info_M, &pBufferSize_M);
	hipsparseDcsrsv2_bufferSize(handle, trans_L, m, nnz, descr_L, d_csrVal, d_csrRowPtr, d_csrColInd, info_L, &pBufferSize_L);
	hipsparseDcsrsv2_bufferSize(handle, trans_U, m, nnz, descr_U, d_csrVal, d_csrRowPtr, d_csrColInd, info_U, &pBufferSize_U);
	pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));
	// pBuffer returned by hipMalloc is automatically aligned to 128 bytes. 
	hipMalloc((void**)&pBuffer, pBufferSize);

	// step 4: perform analysis of incomplete Cholesky on M 
	// perform analysis of triangular solve on L 
	// perform analysis of triangular solve on U
	// The lower(upper) triangular part of M has the same sparsity pattern as L(U), 
	// we can do analysis of csrilu0 and csrsv2 simultaneously.
	checkCudaErrors(hipsparseDcsrilu02_analysis(handle, m, nnz, descr_M, d_csrVal, d_csrRowPtr, d_csrColInd, info_M, policy_M, pBuffer));
	status = hipsparseXcsrilu02_zeroPivot(handle, info_M, &structural_zero);
	if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
		//printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
	}
	hipsparseDcsrsv2_analysis(handle, trans_L, m, nnz, descr_L, d_csrVal, d_csrRowPtr, d_csrColInd, info_L, policy_L, pBuffer);
	hipsparseDcsrsv2_analysis(handle, trans_U, m, nnz, descr_U, d_csrVal, d_csrRowPtr, d_csrColInd, info_U, policy_U, pBuffer);

	// step 5: M = L * U
	//printf("begin to M = L * U\n");
	hipsparseDcsrilu02(handle, m, nnz, descr_M, d_csrVal, d_csrRowPtr, d_csrColInd, info_M, policy_M, pBuffer);
	status = hipsparseXcsrilu02_zeroPivot(handle, info_M, &numerical_zero);
	if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
		//printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero);
	}

	// step 6: solve L*z = x 
	hipsparseDcsrsv2_solve(handle, trans_L,
		m, nnz,
		&alpha, descr_L, d_csrVal, d_csrRowPtr, d_csrColInd, info_L,
		d_x, d_z, policy_L, pBuffer);

	// step 7: solve U*y = z 
	hipsparseDcsrsv2_solve(handle, trans_U,
		m, nnz,
		&alpha, descr_U, d_csrVal, d_csrRowPtr, d_csrColInd, info_U,
		d_z, d_y, policy_U, pBuffer);

	checkCudaErrors(hipMemcpy(res, d_y, m*sizeof(h_y[0]), hipMemcpyDeviceToHost));
	//checkCudaErrors(hipMemcpy(h_y, d_y, m*sizeof(h_y[0]), hipMemcpyDeviceToHost));
	//for (int i = 0; i < m; ++i)
	//	//printf("%f\t", h_y[i]);

	// step 6: free resources 
	hipFree(pBuffer);
	hipsparseDestroyMatDescr(descr_M);
	hipsparseDestroyMatDescr(descr_L);
	hipsparseDestroyMatDescr(descr_U);
	hipsparseDestroyCsrilu02Info(info_M);
	hipsparseDestroyCsrsv2Info(info_L);
	hipsparseDestroyCsrsv2Info(info_U);
	hipsparseDestroy(handle);
}

extern "C" void callKernel_step2_iterative(int *L_rows, int *L_cols, double *L_vals, double *alpha_star, double *res, int m, int nnz)
{

	/* Create CUBLAS context */
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	checkCudaErrors( hipblasCreate(&cublasHandle));

	/* Create CUSPARSE context */
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseStatus_t cusparseStatus;
	checkCudaErrors( hipsparseCreate(&cusparseHandle));

	/* Description of the A matrix*/
	hipsparseMatDescr_t descr = 0;
	checkCudaErrors( hipsparseCreateMatDescr(&descr));

	/* Define the properties of the matrix */
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	//=====================================================================================================//
	//convert from coo format to csr format

	//matrix device
	int * h_csrRowPtr = 0;
	int * csrRowPtr = 0;
	int * cooRowIndex = 0;

	h_csrRowPtr = (int *)malloc((m+1)*sizeof(int));

	/* allocate GPU memory and copy the matrix and vectors into it */
	checkCudaErrors(hipMalloc((void**)&cooRowIndex, nnz*sizeof(cooRowIndex[0])));
	//printf("Device malloc succeeded\n");

	//copy stage
	checkCudaErrors(hipMemcpy(cooRowIndex, L_rows, nnz*sizeof(L_rows[0]), hipMemcpyHostToDevice));
	//printf("Memcpy from Host to Device succeeded\n");

	//change matrix format
	checkCudaErrors(hipMalloc((void**)&csrRowPtr, (m + 1) * sizeof(csrRowPtr[0])));
	checkCudaErrors(hipsparseXcoo2csr(cusparseHandle, cooRowIndex, nnz, m + 1, csrRowPtr, HIPSPARSE_INDEX_BASE_ZERO));
	//printf("Conversion from COO to CSR format succeeded\n");

	checkCudaErrors(hipMemcpy(h_csrRowPtr, csrRowPtr, (m + 1) * sizeof(csrRowPtr[0]), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();

	//===========================================================================================================//

	const int max_iter = 100;		//max iteraton times
	int k, *I = NULL, *J = NULL;
	int *d_col, *d_row;
	const double precision = 1e-10f;
	double *x, *rhs;
	double r0, r1, alpha, beta;
	double *d_val, *d_x;
	double *d_r, *d_p, *d_omega, *d_y;
	double *val = NULL;
	double rsum, diff, err = 0.0;
	double qaerr1, qaerr2 = 0.0;
	double dot, numerator, denominator, nalpha;
	const double doubleone = 1.0;
	const double doublezero = 0.0;

	int nErrors = 0;

	//printf("conjugateGradientPrecond starting...\n");

	/* Generate a random tridiagonal symmetric matrix in CSR (Compressed Sparse Row) format */

	I = (int *)malloc(sizeof(int)*(m + 1));                              // csr row pointers for matrix A
	J = (int *)malloc(sizeof(int)*nnz);                                 // csr column indices for matrix A
	val = (double *)malloc(sizeof(double)*nnz);                           // csr values for matrix A
	x = (double *)malloc(sizeof(double)*m);
	rhs = (double *)malloc(sizeof(double)*m);

	for (int i = 0; i < m; i++)
	{
		rhs[i] = 0.0;                                                  // Initialize RHS
		x[i] = 0.0;                                                    // Initial approximation of solution
	}

	//
	memcpy(I, h_csrRowPtr, (m + 1)*sizeof(int));
	memcpy(J, L_cols, nnz * sizeof(int));
	memcpy(val, L_vals, nnz * sizeof(double));
	memcpy(rhs, alpha_star, m * sizeof(double));
	
	//memcpy(I, h_csrRowPtr, (m+1)*sizeof(int));

	////I = csrRowPtr;
	//J = L_cols;
	//val = L_vals;

	//rhs = alpha_star;
	//
	
	/* Allocate required memory */
	checkCudaErrors(hipMalloc((void **)&d_col, nnz*sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&d_row, (m + 1)*sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&d_val, nnz*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_x, m*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_y, m*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_r, m*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_p, m*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_omega, m*sizeof(double)));

	hipMemcpy(d_col, J, nnz*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row, I, (m + 1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_val, val, nnz*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, m*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_r, rhs, m*sizeof(double), hipMemcpyHostToDevice);

	/* Conjugate gradient without preconditioning.
	------------------------------------------
	Follows the description by Golub & Van Loan, "Matrix Computations 3rd ed.", Section 10.2.6  */

	//printf("Convergence of conjugate gradient without preconditioning: \n");
	k = 0;
	r0 = 0;
	hipblasDdot(cublasHandle, m, d_r, 1, d_r, 1, &r1);

	while (r1 > precision*precision && k <= max_iter)
	{
		k++;

		if (k == 1)
		{
			hipblasDcopy(cublasHandle, m, d_r, 1, d_p, 1);
		}
		else
		{
			beta = r1 / r0;
			hipblasDscal(cublasHandle, m, &beta, d_p, 1);
			hipblasDaxpy(cublasHandle, m, &doubleone, d_r, 1, d_p, 1);
		}

		hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, m, nnz, &doubleone, descr, d_val, d_row, d_col, d_p, &doublezero, d_omega);
		hipblasDdot(cublasHandle, m, d_p, 1, d_omega, 1, &dot);
		alpha = r1 / dot;
		hipblasDaxpy(cublasHandle, m, &alpha, d_p, 1, d_x, 1);
		nalpha = -alpha;
		hipblasDaxpy(cublasHandle, m, &nalpha, d_omega, 1, d_r, 1);
		r0 = r1;
		hipblasDdot(cublasHandle, m, d_r, 1, d_r, 1, &r1);
	}

	//printf("  iteration = %3d, residual = %e \n", k, sqrt(r1));

	hipMemcpy(x, d_x, m*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(res, d_x, m*sizeof(double), hipMemcpyDeviceToHost);

	/* check result */
	err = 0.0;

	for (int i = 0; i < m; i++)
	{
		rsum = 0.0;

		for (int j = I[i]; j < I[i + 1]; j++)
		{
			rsum += val[j] * x[J[j]];
		}

		diff = fabs(rsum - rhs[i]);

		if (diff > err)
		{
			err = diff;
		}
	}

	//printf("  Convergence Test: %s \n", (k <= max_iter) ? "OK" : "FAIL");
	nErrors += (k > max_iter) ? 1 : 0;
	qaerr1 = err;


	/* Destroy contexts */
	hipsparseDestroy(cusparseHandle);
	hipblasDestroy(cublasHandle);

	/* Free device memory */
	free(h_csrRowPtr);
	//free(I);
	//free(J);
	//free(val);
	//free(x);
	//free(rhs);
	//free(valsILU0);
	hipFree(d_col);
	hipFree(d_row);
	hipFree(d_val);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_r);
	hipFree(d_p);
	hipFree(d_omega);

	hipDeviceReset();

	//printf("  Test Summary:\n");
	//printf("     Counted total of %d errors\n", nErrors);
	//printf("     qaerr1 = %f qaerr2 = %f\n\n", fabs(qaerr1), fabs(qaerr2));

}

extern "C" void callKernel_step2_iterative_lu(int *L_rows, int *L_cols, double *L_vals, double *alpha_star, double *res, int m, int nnz)
{

	/* Create CUBLAS context */
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	checkCudaErrors(hipblasCreate(&cublasHandle));

	/* Create CUSPARSE context */
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseStatus_t cusparseStatus;
	checkCudaErrors(hipsparseCreate(&cusparseHandle));

	/* Description of the A matrix*/
	hipsparseMatDescr_t descr = 0;
	checkCudaErrors(hipsparseCreateMatDescr(&descr));

	/* Define the properties of the matrix */
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	//=====================================================================================================//
	//convert from coo format to csr format

	//matrix device
	int * h_csrRowPtr = 0;
	int * csrRowPtr = 0;
	int * cooRowIndex = 0;

	h_csrRowPtr = (int *)malloc((m + 1)*sizeof(int));

	/* allocate GPU memory and copy the matrix and vectors into it */
	checkCudaErrors(hipMalloc((void**)&cooRowIndex, nnz*sizeof(cooRowIndex[0])));
	//printf("Device malloc succeeded\n");

	//copy stage
	checkCudaErrors(hipMemcpy(cooRowIndex, L_rows, nnz*sizeof(L_rows[0]), hipMemcpyHostToDevice));
	//printf("Memcpy from Host to Device succeeded\n");

	//change matrix format
	checkCudaErrors(hipMalloc((void**)&csrRowPtr, (m + 1) * sizeof(csrRowPtr[0])));
	checkCudaErrors(hipsparseXcoo2csr(cusparseHandle, cooRowIndex, nnz, m + 1, csrRowPtr, HIPSPARSE_INDEX_BASE_ZERO));
	//printf("Conversion from COO to CSR format succeeded\n");

	checkCudaErrors(hipMemcpy(h_csrRowPtr, csrRowPtr, (m + 1) * sizeof(csrRowPtr[0]), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();

	//===========================================================================================================//

	const int max_iter = 500;
	int k, *I = NULL, *J = NULL;
	int *d_col, *d_row;
	int qatest = 0;
	const double tol = 1e-1f;
	double *x, *rhs;
	double r0, r1, alpha, beta;
	double *d_val, *d_x;
	double *d_zm1, *d_zm2, *d_rm2;
	double *d_r, *d_p, *d_omega, *d_y;
	double *val = NULL;
	double *d_valsILU0;
	double rsum, diff, err = 0.0;
	double qaerr1, qaerr2 = 0.0;
	double dot, numerator, denominator, nalpha;
	const double doubleone = 1.0;
	const double doublezero = 0.0;

	int nErrors = 0;

	//printf("conjugateGradientPrecond starting...\n");


	I = (int *)malloc(sizeof(int)*(m + 1));                              // csr row pointers for matrix A
	J = (int *)malloc(sizeof(int)*nnz);                                 // csr column indices for matrix A
	val = (double *)malloc(sizeof(double)*nnz);                           // csr values for matrix A
	x = (double *)malloc(sizeof(double)*m);
	rhs = (double *)malloc(sizeof(double)*m);

	for (int i = 0; i < m; i++)
	{
		rhs[i] = 0.0;                                                  // Initialize RHS
		x[i] = 0.0;                                                    // Initial approximation of solution
	}

	//prepareData(I, J, val, M, N, nnz, rhs);
	//printf("data assignment\n");
	
	//I = csrRowPtr;
	//J = L_cols;
	//val = L_vals;
	//rhs = alpha_star;
	memcpy(I, h_csrRowPtr, (m + 1)*sizeof(int));
	memcpy(J, L_cols, nnz * sizeof(int));
	memcpy(val, L_vals, nnz * sizeof(double));
	memcpy(rhs, alpha_star, m * sizeof(double));
	


	/* Allocate required memory */
	checkCudaErrors(hipMalloc((void **)&d_col, nnz*sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&d_row, (m + 1)*sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&d_val, nnz*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_x, m*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_y, m*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_r, m*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_p, m*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_omega, m*sizeof(double)));

	hipMemcpy(d_col, J, nnz*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row, I, (m + 1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_val, val, nnz*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, m*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_r, rhs, m*sizeof(double), hipMemcpyHostToDevice);
	

	/* Preconditioned Conjugate Gradient using ILU.
	--------------------------------------------
	Follows the description by Golub & Van Loan, "Matrix Computations 3rd ed.", Algorithm 10.3.1  */

	//printf("\nConvergence of conjugate gradient using incomplete LU preconditioning: \n");

	int nnzILU0 = nnz;
	//int nnzILU0 = 2*N-1;
	//valsILU0 = (double *) malloc(nnz*sizeof(double));

	checkCudaErrors(hipMalloc((void **)&d_valsILU0, nnz*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_zm1, (m)*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_zm2, (m)*sizeof(double)));
	checkCudaErrors(hipMalloc((void **)&d_rm2, (m)*sizeof(double)));

	/* create the analysis info object for the A matrix */
	cusparseSolveAnalysisInfo_t infoA = 0;
	cusparseStatus = cusparseCreateSolveAnalysisInfo(&infoA);

	checkCudaErrors(cusparseStatus);

	/* Perform the analysis for the Non-Transpose case */
	cusparseStatus = cusparseDcsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		m, nnz, descr, d_val, d_row, d_col, infoA);

	checkCudaErrors(cusparseStatus);

	/* Copy A data to ILU0 vals as input*/
	hipMemcpy(d_valsILU0, d_val, nnz*sizeof(double), hipMemcpyDeviceToDevice);

	/* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
	cusparseStatus = cusparseDcsrilu0(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, descr, d_valsILU0, d_row, d_col, infoA);

	checkCudaErrors(cusparseStatus);

	/* Create info objects for the ILU0 preconditioner */
	cusparseSolveAnalysisInfo_t info_u;
	cusparseCreateSolveAnalysisInfo(&info_u);

	hipsparseMatDescr_t descrL = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrL);
	hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);

	hipsparseMatDescr_t descrU = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrU);
	hipsparseSetMatType(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);

	cusparseStatus = cusparseDcsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, nnz, descrU, d_val, d_row, d_col, info_u);

	/* reset the initial guess of the solution to zero */
	for (int i = 0; i < m; i++)
	{
		x[i] = 0.0;
	}

	checkCudaErrors(hipMemcpy(d_r, rhs, m*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_x, x, m*sizeof(double), hipMemcpyHostToDevice));

	k = 0;
	hipblasDdot(cublasHandle, m, d_r, 1, d_r, 1, &r1);

	while (r1 > tol*tol && k <= max_iter)
	{
		// Forward Solve, we can re-use infoA since the sparsity pattern of A matches that of L
		cusparseStatus = cusparseDcsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, &doubleone, descrL,
			d_valsILU0, d_row, d_col, infoA, d_r, d_y);
		checkCudaErrors(cusparseStatus);

		// Back Substitution
		cusparseStatus = cusparseDcsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, &doubleone, descrU,
			d_valsILU0, d_row, d_col, info_u, d_y, d_zm1);
		checkCudaErrors(cusparseStatus);

		k++;

		if (k == 1)
		{
			hipblasDcopy(cublasHandle, m, d_zm1, 1, d_p, 1);
		}
		else
		{
			hipblasDdot(cublasHandle, m, d_r, 1, d_zm1, 1, &numerator);
			hipblasDdot(cublasHandle, m, d_rm2, 1, d_zm2, 1, &denominator);
			beta = numerator / denominator;
			hipblasDscal(cublasHandle, m, &beta, d_p, 1);
			hipblasDaxpy(cublasHandle, m, &doubleone, d_zm1, 1, d_p, 1);
		}

		hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, m, nnzILU0, &doubleone, descrU, d_val, d_row, d_col, d_p, &doublezero, d_omega);
		hipblasDdot(cublasHandle, m, d_r, 1, d_zm1, 1, &numerator);
		hipblasDdot(cublasHandle, m, d_p, 1, d_omega, 1, &denominator);
		alpha = numerator / denominator;
		hipblasDaxpy(cublasHandle, m, &alpha, d_p, 1, d_x, 1);
		hipblasDcopy(cublasHandle, m, d_r, 1, d_rm2, 1);
		hipblasDcopy(cublasHandle, m, d_zm1, 1, d_zm2, 1);
		nalpha = -alpha;
		hipblasDaxpy(cublasHandle, m, &nalpha, d_omega, 1, d_r, 1);
		hipblasDdot(cublasHandle, m, d_r, 1, d_r, 1, &r1);
	}

	//printf("  iteration = %3d, residual = %e \n", k, sqrt(r1));

	hipMemcpy(x, d_x, m*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(res, d_x, m*sizeof(double), hipMemcpyDeviceToHost);
	/* check result */
	err = 0.0;

	for (int i = 0; i < m; i++)
	{
		rsum = 0.0;

		for (int j = I[i]; j < I[i + 1]; j++)
		{
			rsum += val[j] * x[J[j]];
		}

		diff = fabs(rsum - rhs[i]);

		if (diff > err)
		{
			err = diff;
		}
	}

	//printf("  Convergence Test: %s \n", (k <= max_iter) ? "OK" : "FAIL");
	nErrors += (k > max_iter) ? 1 : 0;
	qaerr2 = err;

	/* Destroy paramters */
	cusparseDestroySolveAnalysisInfo(infoA);
	cusparseDestroySolveAnalysisInfo(info_u);

	/* Destroy contexts */
	hipsparseDestroy(cusparseHandle);
	hipblasDestroy(cublasHandle);

	/* Free device memory */
	free(I);
	free(J);
	free(val);
	free(x);
	free(rhs);
	//free(valsILU0);
	hipFree(d_col);
	hipFree(d_row);
	hipFree(d_val);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_r);
	hipFree(d_p);
	hipFree(d_omega);
	hipFree(d_valsILU0);
	hipFree(d_zm1);
	hipFree(d_zm2);
	hipFree(d_rm2);

	hipDeviceReset();

	//printf("  Test Summary:\n");
	//printf("     Counted total of %d errors\n", nErrors);
	//printf("     qaerr1 = %f qaerr2 = %f\n\n", fabs(qaerr1), fabs(qaerr2));
	//exit((nErrors == 0 && fabs(qaerr1)<1e-5 && fabs(qaerr2) < 1e-5 ? EXIT_SUCCESS : EXIT_FAILURE));

}

//